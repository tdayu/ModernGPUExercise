#include <random>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <array>
#include <assert.h>
#include <algorithm>

#define CUDA_CALL(x) {\
    const hipError_t a=(x);\
    if(a != hipSuccess) {\
        printf(\
            "\nerror in line:%d CUDAError:%s(err_num=%d)\n",\
            __LINE__,\
            hipGetErrorString(a),\
            a);\
        hipDeviceReset();\
        assert(0);\
    }\
}

enum Bound {
    Lower,
    Upper
};


#define FULL_MASK 0xFFFFFFFF
#define WARP_SIZE 32

template<typename T>
void print_array(std::vector<T>& input, unsigned elements_per_row, unsigned limit) {
    unsigned count = 0;
    for (size_t i = 0; i < input.size(); i += elements_per_row){
        printf("%u:", i);
        for (size_t j = 0; j < elements_per_row; j++){
            if (i + j < input.size()) std::cout << "\t" << input[ i + j ];
            count++;
        }
        std::cout << std::endl;
        if (count >= limit) break;
    }
    printf("\n");
}

template<typename T>
void print_comparison(std::vector<T>& CPU, std::vector<T>& GPU, unsigned elements_per_row, unsigned limit){
    assert(CPU.size() == GPU.size());

    unsigned count = 0;
    for (size_t i = 0; i < CPU.size(); i += elements_per_row){
        printf("CPU %u:", i);
        for (size_t j = 0; j < elements_per_row; j++){
            if (i + j < CPU.size()) std::cout << "\t" << CPU[ i + j ];
            count++;
        }
        printf("\n");
        printf("GPU %u:", i);
        for (size_t j = 0; j < elements_per_row; j++){
            if (i + j < GPU.size()) std::cout << "\t" << GPU[ i + j ];
        }
        printf("\n\n");
        if (count >= limit) break;
    }
    printf("\n");
}

template<typename T>
void print_if_mismatch(std::vector<T>& CPU, std::vector<T>& GPU, unsigned limit){
    assert(CPU.size() == GPU.size());

    unsigned count = 0;
    for (size_t i = 0; i < CPU.size(); i++){
        if (CPU[i] != GPU[i]){
            printf("Values mismatch! Index: %lu, CPU: %u, GPU: %u, Previous CPU: %u, Previous GPU: %u\n", i, CPU[i], GPU[i], CPU[i-1], GPU[i-1]);
            count++;
        }
        if (count >= limit) break;
    }
}

__device__ unsigned int laneid()
{
    unsigned int laneid;
    asm ("mov.u32 %0, %%laneid;" : "=r"(laneid));
    return laneid;
}

template<typename T, unsigned NT, unsigned VT>
__device__ void global_to_shared(const unsigned total, const T * global, T * shared){
    constexpr unsigned NV = NT * VT;
    #pragma unroll
    for (int i = 0; i < VT; i++){
        const unsigned global_index = NV * blockIdx.x + NT * i + threadIdx.x;
        const unsigned shared_index = NT * i + threadIdx.x;
        shared[shared_index] = (global_index < total) ? global[global_index] : 0;
    }
    __syncthreads();
}

template<typename T, unsigned VT>
__device__ void shared_to_registers(const T * shared, T * registers){
    #pragma unroll
    for (int i = 0; i < VT; i++){
        registers[i] = shared[VT * threadIdx.x + i];
    }
}

template<typename T, unsigned NT, unsigned VT>
__device__ void shared_to_global(const unsigned total, T * global, T * shared){
    constexpr unsigned NV = NT * VT;
    #pragma unroll
    for (int i = 0; i < VT; i++){
        const unsigned global_index = NV * blockIdx.x + NT * i + threadIdx.x;
        const unsigned shared_index = NT * i + threadIdx.x;
        if (global_index < total) global[global_index] = shared[shared_index];
    }
}

template<typename T, unsigned VT>
__device__ void registers_to_shared(T * shared, const T * registers){
    #pragma unroll
    for (int i = 0; i < VT; i++){
        shared[VT * threadIdx.x + i] = registers[i];
    }
    __syncthreads();
}

// NT = number of threads
// VT = values per thread
// NV = number of values per block
template <Bound bound, typename T, typename COMP>
__device__ int merge_path(
    const T * listA,
    const T * listB,
    const int countA,
    const int countB,
    const int diagonal,
    COMP comp){

    int a_begin = max(0, diagonal - countB);
    int a_end = min(diagonal, countA);

    while (a_begin < a_end){
        int a_mid = (a_begin + a_end) / 2;
        T aKey = listA[a_mid];
        T bKey = listB[diagonal - a_mid - 1];

        bool predicate = (bound == Bound::Lower) ? !comp(bKey, aKey) : comp(aKey, bKey);

        if (predicate) a_begin = a_mid + 1;
        else a_end = a_mid;
    }

    return a_begin;
}

// NT = number of threads
// VT = values per thread
// NV = number of values per block
template <Bound bound, typename T, typename COMP, unsigned NT>
__global__ void merge_path_partition(
    const T * listA,
    const T * listB,
    const int countA,
    const int countB,
    const int partition_size,
    const int number_of_partitions,
    COMP comp,
    int * partitions){

    const int total = countA + countB;
    const int diagonal = min((blockIdx.x * NT + threadIdx.x) * partition_size, total);

    if ((blockIdx.x * NT + threadIdx.x) <= number_of_partitions){
        const int a_begin = merge_path<bound>(listA, listB, countA, countB, diagonal, comp);
        partitions[blockIdx.x * NT + threadIdx.x] = a_begin;
    }
}

// NT = number of threads
// VT = values per thread
// NV = number of values per block
template <Bound bound, unsigned VT, typename T, typename COMP>
__device__ void serial_sorted_search(
    const T * keysA,
    const T * keysB,
    const int beginA,
    const int countA,
    const int countB,
    const int diagonal,
    COMP comp,
    unsigned * indicesA,
    unsigned * indicesB){

    const int beginB = diagonal - beginA;
    bool in_rangeA = beginA < countA;
    bool in_rangeB = beginB < countB;

    T keyA, keyB;
    if (in_rangeA) keyA = keysA[beginA];
    if (in_rangeB) keyB = keysB[beginB];

    unsigned indexA = beginA;
    unsigned indexB = diagonal - indexA;

    #pragma unroll
    for (unsigned i = 0; i < VT; i++){
        if ( indexA < countA || indexB < countB ){
            bool predicate;
            if (indexB >= countB) predicate = true;
            else if (indexA >= countA) predicate = false;
            else predicate = (bound == Bound::Lower) ?
                             !comp(keysB[indexB], keysA[indexA]) :
                             comp(keysA[indexA], keysB[indexB]);
            if (predicate){
                indicesA[indexA++] = indexB;
            }
            else{
                indicesB[indexB++] = indexA;
            }
        }
    }
}

// NT = number of threads
// VT = values per thread
// NV = number of values per block
template<Bound bound, typename T, typename COMP, unsigned NT, unsigned VT>
__global__ void block_vectorized_sorted_search(
    const T * listA,
    const T * listB,
    const int countA,
    const int countB,
    const int * partitions,
    COMP comp,
    unsigned * outputA,
    unsigned * outputB){

    constexpr unsigned NV = NT * VT;
    // union Shared {
    //     T keys[NV];
    //     unsigned indices[NV];
    // };
    // __shared__ Shared shared;
    __shared__ T shared_keys[NV];
    __shared__ unsigned shared_indices[NV];

    const int total = countA + countB;
    const int block_total = min(NV, total - blockIdx.x * NV);
    const int block_countA = partitions[blockIdx.x + 1] - partitions[blockIdx.x];
    const int block_countB = block_total - block_countA;

    const int offsetA = partitions[blockIdx.x];
    const int offsetB = blockIdx.x * NV - offsetA;

    #pragma unroll
    for (unsigned i = 0; i < VT; i++){
        const unsigned index = NT * i + threadIdx.x;
        if (index < block_total) {
            shared_keys[index] = (index < block_countA) ? 
                                 listA[offsetA + index] :
                                 listB[offsetB + index - block_countA];
        }
    }
    __syncthreads();

    const int diagonal = min(threadIdx.x * VT, block_total);
    const int beginA = merge_path<bound>(shared_keys, shared_keys + block_countA, block_countA, block_countB, diagonal, comp);
    __syncthreads();

    serial_sorted_search<bound, VT>(
        shared_keys,
        shared_keys + block_countA,
        beginA,
        block_countA,
        block_countB,
        diagonal,
        comp,
        shared_indices,
        shared_indices + block_countA);
    __syncthreads();

    #pragma unroll
    for (unsigned i = 0; i < VT; i++){
        if (blockIdx.x * NV + i * NT + threadIdx.x < total) {
            if (i * NT + threadIdx.x < block_countA){
                outputA[offsetA + i * NT + threadIdx.x] = offsetB + shared_indices[i * NT + threadIdx.x];
            }
            else{
                outputB[offsetB + i * NT + threadIdx.x - block_countA] = offsetA + shared_indices[i * NT + threadIdx.x];
            }
        }
    }
}

template<typename T>
class leq{
    public:
        __host__ __device__
        bool operator()(const T a, const T b){
            return a <= b;
        };
};

template<typename T>
class lt{
    public:
        __host__ __device__
        bool operator()(const T a, const T b){
            return a < b;
        };
};

// NT = number of threads
// VT = values per thread
// NV = number of values per block
template <Bound bound, unsigned NT, unsigned VT, unsigned PartitionNT, typename T, typename COMP>
void gpu_vectorized_sorted_search(
    const std::vector<T>& host_listA,
    const std::vector<T>& host_listB,
    const int countA,
    const int countB,
    std::vector<unsigned>& host_outputA,
    std::vector<unsigned>& host_outputB,
    COMP comp)
{
    constexpr unsigned NV = NT * VT;
    const int total = countA + countB;
    const int number_of_partitions = ( total + NV - 1 ) / NV;
    const int partition_number_of_blocks = ( number_of_partitions / PartitionNT ) + 1;

    T * dev_listA;
    T * dev_listB;
    T * dev_outputA;
    T * dev_outputB;
    int * dev_partitions;

    CUDA_CALL(hipSetDevice(0));
    CUDA_CALL(hipMalloc(&dev_listA, countA * sizeof(T)));
    CUDA_CALL(hipMalloc(&dev_listB, countB * sizeof(T)));
    CUDA_CALL(hipMalloc(&dev_outputA, total * sizeof(unsigned)));
    CUDA_CALL(hipMalloc(&dev_outputB, total * sizeof(unsigned)));
    CUDA_CALL(hipMalloc(&dev_partitions, (number_of_partitions + 1) * sizeof(int)));

    CUDA_CALL(hipMemcpy(dev_listA, host_listA.data(), countA * sizeof(T), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dev_listB, host_listB.data(), countB * sizeof(T), hipMemcpyHostToDevice));

    merge_path_partition<bound, T, COMP, PartitionNT><<<partition_number_of_blocks, PartitionNT>>>(
        dev_listA,
        dev_listB,
        countA,
        countB,
        NV,
        number_of_partitions,
        comp,
        dev_partitions);

    block_vectorized_sorted_search<bound, T, COMP, NT, VT><<<number_of_partitions, NT>>>(
        dev_listA,
        dev_listB,
        countA,
        countB,
        dev_partitions,
        comp,
        dev_outputA,
        dev_outputB);

    CUDA_CALL(hipMemcpy(host_outputA.data(), dev_outputA, countA * sizeof(T), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(host_outputB.data(), dev_outputB, countB * sizeof(T), hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(dev_listA));
    CUDA_CALL(hipFree(dev_listB));
    CUDA_CALL(hipFree(dev_outputA));
    CUDA_CALL(hipFree(dev_outputB));
    CUDA_CALL(hipFree(dev_partitions));
}

int main(int argc, char** argv) {
    std::mt19937 gen(1234);
    std::uniform_int_distribution<> prefix_sum_distribution_A(0, 16);
    std::uniform_int_distribution<> prefix_sum_distribution_B(0, 4);

    constexpr unsigned NT = 256;
    constexpr unsigned VT = 7;
    constexpr unsigned PartitionNT = 128;

    // constexpr std::array<unsigned, 4> NT_values = { 128, 256, 512, 1024 };
    // constexpr std::array<unsigned, 8> VT_values = { 1, 2, 3, 4, 5, 6, 7, 8 };

    const size_t countA = (argc == 3) ? std::stol(argv[1]) : 1024 * 256;
    const size_t countB = (argc == 3) ? std::stol(argv[2]) : 1024 * 1024;
    // const size_t total = countA + countB;
    std::vector<unsigned> host_listA(countA);
    std::vector<unsigned> host_listB(countB);
    std::vector<unsigned> host_cpu_lower_boundA(countA);
    std::vector<unsigned> host_cpu_upper_boundA(countA);
    std::vector<unsigned> host_cpu_lower_boundB(countB);
    std::vector<unsigned> host_cpu_upper_boundB(countB);
    std::vector<unsigned> host_lower_boundA(countA);
    std::vector<unsigned> host_upper_boundA(countA);
    std::vector<unsigned> host_lower_boundB(countB);
    std::vector<unsigned> host_upper_boundB(countB);

    for (size_t i = 0; i < countA; i++){
        host_listA[i] = prefix_sum_distribution_A(gen);
    }
    for (size_t i = 0; i < countB; i++){
        host_listB[i] = prefix_sum_distribution_B(gen);
    }

    std::inclusive_scan(host_listA.begin(), host_listA.end(), host_listA.begin());
    std::inclusive_scan(host_listB.begin(), host_listB.end(), host_listB.begin());

    lt<unsigned> comp;

    // lower bound of A into B
    unsigned indexA = 0;
    unsigned indexB = 0;

    unsigned * listA = host_listA.data();
    unsigned * listB = host_listB.data();

    while ( indexA < countA || indexB < countB ){
        bool predicate;
        if (indexB >= countB) predicate = true;
        else if (indexA >= countA) predicate = false;
        else predicate = !comp(listB[indexB], listA[indexA]);

        if (predicate){
            host_cpu_lower_boundA[indexA++] = indexB;
        }
        else{
            host_cpu_upper_boundB[indexB++] = indexA;
        }
    }

    // upper bound of A into B
    indexA = 0;
    indexB = 0;
    while ( indexA < countA || indexB < countB ){
        bool predicate;
        if (indexB >= countB) predicate = true;
        else if (indexA >= countA) predicate = false;
        else predicate = comp(listA[indexA], listB[indexB]);

        if (predicate){
            host_cpu_upper_boundA[indexA++] = indexB;
        }
        else{
            host_cpu_lower_boundB[indexB++] = indexA;
        }
    }

    gpu_vectorized_sorted_search<Bound::Lower, NT, VT, PartitionNT>(
        host_listA,
        host_listB,
        countA,
        countB,
        host_lower_boundA,
        host_upper_boundB,
        comp);

    gpu_vectorized_sorted_search<Bound::Upper, NT, VT, PartitionNT>(
        host_listA,
        host_listB,
        countA,
        countB,
        host_upper_boundA,
        host_lower_boundB,
        comp);

    printf("ListA:\n");
    print_array(host_listA, 10, 100);
    printf("ListB:\n");
    print_array(host_listB, 10, 100);

    printf("A lower bound:\n");
    print_array(host_cpu_lower_boundA, 10, 100);
    printf("B upper bound:\n");
    print_array(host_cpu_upper_boundB, 10, 100);

    printf("A upper bound:\n");
    print_array(host_cpu_upper_boundA, 10, 100);
    printf("B lower bound:\n");
    print_array(host_cpu_lower_boundB, 10, 100);

    printf("A lower bound:\n");
    print_comparison(host_cpu_lower_boundA, host_lower_boundA, 10, 100);
    printf("B upper bound:\n");
    print_comparison(host_cpu_upper_boundB, host_upper_boundB, 10, 100);
    printf("A upper bound:\n");
    print_comparison(host_cpu_upper_boundA, host_upper_boundA, 10, 100);
    printf("B lower bound:\n");
    print_comparison(host_cpu_lower_boundB, host_lower_boundB, 10, 100);

    printf("A lower bound mismatches:\n");
    print_if_mismatch(host_cpu_lower_boundA, host_lower_boundA, 100);
    printf("B upper bound mismatches:\n");
    print_if_mismatch(host_cpu_upper_boundB, host_upper_boundB, 100);
    printf("A upper bound mismatches:\n");
    print_if_mismatch(host_cpu_upper_boundA, host_upper_boundA, 100);
    printf("B lower bound mismatches:\n");
    print_if_mismatch(host_cpu_lower_boundB, host_lower_boundB, 100);

    return 0;
    // constexpr lt<unsigned> comp;

    // gpu_merge<NT, VT, PartitionNT>(
    //     host_listA,
    //     host_listB,
    //     int(countA),
    //     int(countB),
    //     host_output,
    //     comp
    // );

    // print_if_mismatch(host_cpu_merged, host_output, 400);
}
